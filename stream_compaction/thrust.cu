#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction
{
  namespace Thrust
  {
    using StreamCompaction::Common::PerformanceTimer;

    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    /**
     * Performs prefix-sum (aka scan) on idata, storing the result into odata.
     */
    void scan(int n, int* odata, const int* idata)
    {
      // TODO use `thrust::exclusive_scan`
      // example: for device_vectors dv_in and dv_out:
      // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

      thrust::host_vector<int> thrust_host_idata = thrust::host_vector<int>(n);

      for (int idx = 0; idx < n; ++idx)
      {
        thrust_host_idata[idx] = idata[idx];
      }

      thrust::device_vector<int> thrust_device_idata = thrust_host_idata;
      thrust::device_vector<int> thrust_device_odata = thrust::device_vector<int>(n, 0);
      
      timer().startGpuTimer();
      thrust::exclusive_scan(thrust_device_idata.begin(), thrust_device_idata.end(), thrust_device_odata.begin());
      timer().endGpuTimer();

      thrust::copy(thrust_device_odata.begin(), thrust_device_odata.end(), odata);
    }
  }
}
