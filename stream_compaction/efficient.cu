#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "naive.h"

#define blockSize 256

namespace StreamCompaction
{
  namespace Efficient
  {
    int* device_idata;
    int* device_bools;
    int* device_scannedBools;
    int* device_odata;
    int numObjects;

    using StreamCompaction::Common::PerformanceTimer;

    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    __global__ void kernel_UpSweep(int N, int powDP1, int* idata)
    {
      const int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= N) {
        return;
      }

      if (index % powDP1 != 0)
      {
        return;
      }

      // x[k + 2d+1 � 1] += x[k + 2d � 1];
      idata[index + powDP1 - 1] += idata[index + (powDP1 / 2) - 1];
    }

    __global__ void kernel_DownSweep(int N, int powDP1, int* idata)
    {
      const int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= N) {
        return;
      }

      if (index % powDP1 != 0)
      {
        return;
      }

      // Calculate some indices
      const int leftChildIdx = index + (powDP1 / 2) - 1;
      const int rightChildIdx = index + powDP1 - 1;

      // Save the left child
      const int leftChild = idata[leftChildIdx];

      // Set Left Child to Current Node's Value
      idata[leftChildIdx] = idata[rightChildIdx];

      // Set Right Child to Left + Right
      idata[rightChildIdx] += leftChild;
    }

    /**
     * Performs prefix-sum (aka scan) on idata, storing the result into odata.
     */
    void scan(int n, int* odata, const int* idata)
    {
      numObjects = n;
      const int logN = ilog2ceil(numObjects);
      const int nearestPower2 = std::pow(2, logN);

      hipMalloc((void**)&device_idata, nearestPower2 * sizeof(int));
      checkCUDAError("hipMalloc device_idata failed!");

      hipMemcpy(device_idata, idata, sizeof(int) * nearestPower2, hipMemcpyHostToDevice);

      const int numBlocks = (numObjects + blockSize - 1) / blockSize;
      dim3 fullBlocksPerGrid(numBlocks);

      int* loopInputBuffer = device_idata;

      // Up Sweep
      timer().startGpuTimer();
      for (int d = 0; d < logN; ++d)
      {
        const int powDP1 = std::pow(2, d + 1);
        kernel_UpSweep<<<fullBlocksPerGrid, blockSize>>>(numObjects, powDP1, loopInputBuffer);
      }
      timer().endGpuTimer();

      // Set x[n-1] = 0
      // This seems really weird that we need to copy a 0 from host to the device.
      // Might need to find a more efficient way.
      const int lastValue = 0;
      hipMemcpy(&loopInputBuffer[nearestPower2 - 1], &lastValue, sizeof(int), hipMemcpyHostToDevice);

      // Down Sweep
      timer().startGpuTimer();
      for (int d = logN - 1; d >= 0; --d)
      {
        const int powDP1 = std::pow(2, d + 1);
        kernel_DownSweep<<<fullBlocksPerGrid, blockSize>>>(numObjects, powDP1, loopInputBuffer);
      }
      timer().endGpuTimer();

      hipMemcpy(odata, loopInputBuffer, sizeof(int) * (numObjects), hipMemcpyDeviceToHost);

      hipFree(device_idata);
    }

    /**
     * Performs stream compaction on idata, storing the result into odata.
     * All zeroes are discarded.
     *
     * @param n      The number of elements in idata.
     * @param odata  The array into which to store elements.
     * @param idata  The array of elements to compact.
     * @returns      The number of elements remaining after compaction.
     */
    int compact(int n, int* odata, const int* idata)
    {
      numObjects = n;
      const int logN = ilog2ceil(numObjects);
      const int nearestPower2 = std::pow(2, logN);

      hipMalloc((void**)&device_idata, nearestPower2 * sizeof(int));
      checkCUDAError("hipMalloc device_idata failed!");
      
      hipMalloc((void**)&device_odata, nearestPower2 * sizeof(int));
      checkCUDAError("hipMalloc device_odata failed!");

      hipMalloc((void**)&device_bools, nearestPower2 * sizeof(int));
      checkCUDAError("hipMalloc device_bools failed!");
      
      hipMalloc((void**)&device_scannedBools, nearestPower2 * sizeof(int));
      checkCUDAError("hipMalloc device_scannedBools failed!");

      hipMemcpy(device_idata, idata, sizeof(int) * nearestPower2, hipMemcpyHostToDevice);

      const int numBlocks = (numObjects + blockSize - 1) / blockSize;
      dim3 fullBlocksPerGrid(numBlocks);

      // 1. Get Bool Array 1st
      timer().startGpuTimer();
      Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(numObjects, device_bools, device_idata);
      timer().endGpuTimer();

      hipMemcpy(device_scannedBools, device_bools, sizeof(int) * nearestPower2, hipMemcpyDeviceToDevice);

      // 2. Scan the Bool Array
      int* loopInputBuffer = device_scannedBools;

      // Up Sweep
      timer().startGpuTimer();
      for (int d = 0; d < logN; ++d)
      {
        const int powDP1 = std::pow(2, d + 1);
        kernel_UpSweep<<<fullBlocksPerGrid, blockSize>>>(numObjects, powDP1, loopInputBuffer);
      }
      timer().endGpuTimer();

      // Set x[n-1] = 0
      // This seems really weird that we need to copy a 0 from host to the device.
      // Might need to find a more efficient way.
      const int lastValue = 0;
      hipMemcpy(&loopInputBuffer[nearestPower2 - 1], &lastValue, sizeof(int), hipMemcpyHostToDevice);

      // Down Sweep
      timer().startGpuTimer();
      for (int d = logN - 1; d >= 0; --d)
      {
        const int powDP1 = std::pow(2, d + 1);
        kernel_DownSweep<<<fullBlocksPerGrid, blockSize>>>(numObjects, powDP1, loopInputBuffer);
      }
      timer().endGpuTimer();

      // 3. Store in OData
      Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(numObjects, device_odata, device_idata, device_bools, device_scannedBools);

      int boolArrayLast = 0;
      hipMemcpy(&boolArrayLast, &device_bools[nearestPower2 - 1], sizeof(int), hipMemcpyDeviceToHost);

      int scannedLast = 0;
      hipMemcpy(&scannedLast, &device_scannedBools[nearestPower2 - 1], sizeof(int), hipMemcpyDeviceToHost);

      const int totalEntries = scannedLast + boolArrayLast;
      hipMemcpy(odata, device_odata, sizeof(int) * (totalEntries), hipMemcpyDeviceToHost);

      hipFree(device_idata);
      hipFree(device_odata);
      hipFree(device_bools);
      hipFree(device_scannedBools);
      return totalEntries;
    }
  }
}
